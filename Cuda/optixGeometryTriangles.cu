#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

struct Vertex {
    float3 position;
    float3 normal;
    float2 texCoord;
};


rtDeclareVariable( float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float2, texcoord,         attribute texcoord, );
rtDeclareVariable( float2, barycentrics,     attribute barycentrics, );

//rtBuffer<float3> vertex_buffer;
//rtBuffer<float3> normal_buffer;
//rtBuffer<float2> texcoord_buffer;
rtBuffer<uint3,1>   index_buffer;

rtBuffer<Vertex,1> vertex_buffer;

RT_PROGRAM void triangle_attributes()
{
    const uint3   v_idx = index_buffer[rtGetPrimitiveIndex()];
    //const float3 v0    = vertex_buffer[v_idx.x];
    const Vertex v0 = vertex_buffer[v_idx.x];
    //const float3 v1    = vertex_buffer[v_idx.y];
    const Vertex v1 = vertex_buffer[v_idx.y];
    //const float3 v2    = vertex_buffer[v_idx.z];
    const Vertex v2 = vertex_buffer[v_idx.z];
    const float3 Ng    = optix::cross( v1.position - v0.position, v2.position - v0.position );

    geometric_normal = optix::normalize( Ng );

    barycentrics = rtGetTriangleBarycentrics();
    //texcoord = make_float3( barycentrics.x, barycentrics.y, 0.0f );

    shading_normal = v1.normal * barycentrics.x + v2.normal * barycentrics.y
            + v0.normal * ( 1.0f-barycentrics.x-barycentrics.y );
    
    const float2 t0 = v0.texCoord;
    const float2 t1 = v1.texCoord;
    const float2 t2 = v2.texCoord;

    texcoord = t1*barycentrics.x + t2*barycentrics.y + t0*(1.0f-barycentrics.x-barycentrics.y);
}
