#include "hip/hip_runtime.h"
#include "optixSVGF.h"
#include "../common.h"
#include "../random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    float3 direct_radiance;
    float3 indirect_radiance;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,         launch_dim,   rtLaunchDim, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

//rtDeclareVariable(float,         focal_length, , );
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>              output_direct_buffer;
rtBuffer<float4, 2>              output_indirect_buffer;

rtBuffer<ParallelogramLight>     lights;

rtBuffer<float4, 2>              direct_color_buffer;
rtBuffer<float4, 2>              indirect_color_buffer;
rtBuffer<float4, 2>              color_buffer;


// RT_PROGRAM void pathtrace_camera()
// {
//     size_t2 screen = output_buffer.size();

//     float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
//     float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

//     float2 jitter_scale = inv_screen / sqrt_num_samples;
//     unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

//     float3 result = make_float3(0.0f);
//     float3 result_direct = make_float3(0.0f);
//     float3 result_indirect = make_float3(0.0f);

//     unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
//     while(samples_per_pixel--)
//     {
//         //
//         // Sample pixel using jittering
//         //
//         unsigned int x = samples_per_pixel%sqrt_num_samples;
      
//         unsigned int y = samples_per_pixel/sqrt_num_samples;
//         //printf("samples %u xy %u %u\n\n", samples_per_pixel, x, y);
//         //printf("Hello from index %u, %u!\n", launch_index.x, launch_index.y);
//         //printf("jitter %f\n%f\n\n",x - rnd(seed), y - rnd(seed));
//         //float2 jitter = make_float2(x+rnd(seed), y+rnd(seed));
//         float2 jitter = make_float2(x+0.5, y+0.5);
        
//         float2 d = pixel + jitter*jitter_scale;
//         float3 ray_origin = eye;
//         float3 ray_direction = normalize(d.x*U + d.y*V + W);

//         // Initialze per-ray data
//         PerRayData_pathtrace prd;
//         prd.result = make_float3(0.f);
//         prd.attenuation = make_float3(1.f);
//         prd.countEmitted = true;
//         prd.done = false;
//         prd.seed = seed;
//         prd.depth = 0;

//         // Each iteration is a segment of the ray path.  The closest hit will
//         // return new segments to be traced here.
//         for(;;)
//         {
//             Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
//             rtTrace(top_object, ray, prd);
          
//             if (prd.depth == 0) {
//                 prd.direct_radiance = prd.radiance * prd.attenuation;
//                 //printf("radiance : %f %f %f   attenuation : %f %f %f\n", prd.radiance.x, prd.radiance.y, prd.radiance.z, prd.attenuation.x, prd.attenuation.y, prd.attenuation.z);
//             }

//             if(prd.done)
//             {
//                 // We have hit the background or a luminaire
//                 prd.result += prd.radiance * prd.attenuation;
//                 break;
//             }

//             // Russian roulette termination 
//             if(prd.depth >= rr_begin_depth)
//             {
//                 float pcont = fmaxf(prd.attenuation);
//                 if(rnd(prd.seed) >= pcont)
//                     break;
//                 prd.attenuation /= pcont;
//             }

//             prd.depth++;
//             prd.result += prd.radiance * prd.attenuation;

//             //printf("radiance : %f %f %f   attenuation : %f %f %f\n", prd.radiance.x, prd.radiance.y, prd.radiance.z, prd.attenuation.x, prd.attenuation.y, prd.attenuation.z);

//             // Update ray data for the next path segment
//             ray_origin = prd.origin;
//             ray_direction = prd.direction;
//         }

//         prd.indirect_radiance = prd.result - prd.direct_radiance;

//         result += prd.result;
//         result_direct += prd.direct_radiance;
//         result_indirect += prd.indirect_radiance;
//         seed = prd.seed;
//     }

//     //
//     // Update the output buffer
//     //
//     float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
//     float3 pixel_color_direct = result_direct / (sqrt_num_samples * sqrt_num_samples);
//     float3 pixel_color_indirect = result_indirect / (sqrt_num_samples * sqrt_num_samples);

//     direct_color_buffer[launch_index] = make_float4(pixel_color_direct, 1.0f);
//     indirect_color_buffer[launch_index] = make_float4(pixel_color_indirect, 1.0f);
//     color_buffer[launch_index] = make_float4(pixel_color, 1.0f);
//     //printf("%f\n", pixel_color);

//     /*if (frame_number > 1)
//     {
//         float a = 1.0f / (float)frame_number;
//         float3 old_color = make_float3(output_buffer[launch_index]);
//         float3 old_direct_color = make_float3(output_direct_buffer[launch_index]);
//         float3 old_indirect_color = make_float3(output_indirect_buffer[launch_index]);

//         output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
//         output_direct_buffer[launch_index] = make_float4(lerp(old_direct_color, pixel_color_direct, a), 1.0f);
//         output_indirect_buffer[launch_index] = make_float4(lerp(old_indirect_color, pixel_color_indirect, a), 1.0f);
//     }
//     else
//     {
//         output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
//         output_direct_buffer[launch_index] = make_float4(pixel_color_direct, 1.0f);
//         output_indirect_buffer[launch_index] = make_float4(pixel_color_indirect, 1.0f);
//     }*/
// }

// rtBuffer<float3, 2>              world_pos_buffer;
// rtBuffer<float3, 2>              world_normal_buffer;
// rtBuffer<float3, 2>              albedo_buffer;

rtTextureSampler<float4, 2>              world_pos_tex;
rtTextureSampler<float4, 2>              world_normal_tex;
rtTextureSampler<float4, 2>              albedo_tex;

RT_PROGRAM void pathtrace_Gbuffer()
{
    size_t2 screen = launch_dim;
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number);

    float2 texCd = (make_float2(launch_index) + 0.5) / make_float2(screen);

    float3 ray_origin = make_float3(tex2D(world_pos_tex, texCd.x, texCd.y));
    float3 ffnormal = make_float3(tex2D(world_normal_tex, texCd.x, texCd.y));
    float3 diffuse_color = make_float3(tex2D(albedo_tex, texCd.x, texCd.y));

    float z1 = rnd(seed);
    float z2 = rnd(seed);
    float3 ray_direction;
    cosine_sample_hemisphere(z1, z2, ray_direction);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( ray_direction );

    // Initialze per-ray data
    PerRayData_pathtrace prd;
    prd.result = make_float3(0.f);
    prd.attenuation = diffuse_color;
    prd.countEmitted = false;
    prd.done = false;
    prd.seed = seed;
    prd.depth = 0;

    unsigned int num_lights = lights.size();
    float3 lightRadiance = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(prd.seed);
        const float z2 = rnd(prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - ray_origin);
        const float3 L     = normalize(light_pos - ray_origin);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( ray_origin, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                lightRadiance += light.emission * weight;
            }
        }
    }
    prd.radiance = lightRadiance;
    prd.result = prd.radiance * prd.attenuation;

    prd.direct_radiance = prd.radiance * prd.attenuation;


    // Each iteration is a segment of the ray path.  The closest hit will
    // return new segments to be traced here.
    for (;;)
    {
        Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

        // if (prd.depth == 0) {
        //     prd.direct_radiance = prd.radiance * prd.attenuation;
        //     //printf("radiance : %f %f %f   attenuation : %f %f %f\n", prd.radiance.x, prd.radiance.y, prd.radiance.z, prd.attenuation.x, prd.attenuation.y, prd.attenuation.z);
        // }
        prd.depth++;
        if (prd.done)
        {
            // We have hit the background or a luminaire
            prd.result += prd.radiance * prd.attenuation;
            break;
        }

        // Russian roulette termination 
        if (prd.depth >= rr_begin_depth)
        {
            float pcont = fmaxf(prd.attenuation);
            if (rnd(prd.seed) >= pcont)
                break;
            prd.attenuation /= pcont;
        }

        //prd.depth++;
        prd.result += prd.radiance * prd.attenuation;

        //printf("radiance : %f %f %f   attenuation : %f %f %f\n", prd.radiance.x, prd.radiance.y, prd.radiance.z, prd.attenuation.x, prd.attenuation.y, prd.attenuation.z);

        // Update ray data for the next path segment
        ray_origin = prd.origin;
        ray_direction = prd.direction;
    }

    prd.indirect_radiance = prd.result - prd.direct_radiance;

    float3 result = prd.result;
    float3 result_direct = prd.direct_radiance;
    float3 result_indirect = prd.indirect_radiance;
    
    float3 pixel_color = result;
    float3 pixel_color_direct = result_direct;
    float3 pixel_color_indirect = result_indirect;

    direct_color_buffer[launch_index] = make_float4(pixel_color_direct, 1.0f);
    indirect_color_buffer[launch_index] = make_float4(pixel_color_indirect, 1.0f);
    color_buffer[launch_index] = make_float4(pixel_color, 1.0f);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        float3 old_direct_color = make_float3(output_direct_buffer[launch_index]);
        float3 old_indirect_color = make_float3(output_indirect_buffer[launch_index]);

        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
        output_direct_buffer[launch_index] = make_float4(lerp(old_direct_color, pixel_color_direct, a), 1.0f);
        output_indirect_buffer[launch_index] = make_float4(lerp(old_indirect_color, pixel_color_indirect, a), 1.0f);
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
        output_direct_buffer[launch_index] = make_float4(pixel_color_direct, 1.0f);
        output_indirect_buffer[launch_index] = make_float4(pixel_color_indirect, 1.0f);
    }
}


rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void closest_hit_emitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


rtDeclareVariable(float3,       Kd, , );
rtTextureSampler<float4, 2>   diffuse_map1;

rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(float2,     texcoord,         attribute texcoord, ); 
rtDeclareVariable(uint,       mesh_id,          attribute mesh_id, );

rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );


RT_PROGRAM void closest_hit()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    float3 diffuse_color = make_float3(tex2D(diffuse_map1, texcoord.x, texcoord.y));
    //float3 diffuse_color = make_float3(1.0f);
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon );
            //top_shadower is not used.
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}



rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

RT_PROGRAM void any_hit()
{
    if (tex2D(diffuse_map1, texcoord.x, texcoord.y).w < 0.1f) {
        rtIgnoreIntersection();
    } 
}



RT_PROGRAM void exception()
{
    printf("optix: gg");
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}



rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
    current_prd.done = true;
}


