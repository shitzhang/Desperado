#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "phong.h"

using namespace optix;


rtDeclareVariable(float3,       Kd, , );
rtTextureSampler<float4, 2> texture_diffuse1;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float2, texcoord, attribute texcoord, ); 

rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = optix::make_float3(0.0f);
	rtTerminateRay();
}

RT_PROGRAM void any_hit() {
  if (tex2D(texture_diffuse1, texcoord.x, texcoord.y).w < 0.1f) {
    rtIgnoreIntersection();
  } 
}

RT_PROGRAM void closest_hit()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 Kd_val = make_float3(tex2D( texture_diffuse1, texcoord.x, texcoord.y ));
  //float3 Kd_val = make_float3(tex2D( texture_diffuse1, 0.5f, 0.5f ));
  prd.result = Kd_val;
  //printf("color %f %f %f   ",prd.result.x,prd.result.y,prd.result.z);
}


