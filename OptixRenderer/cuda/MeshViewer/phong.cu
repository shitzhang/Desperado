#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "phong.h"

using namespace optix;


rtDeclareVariable(float3,       Kd, , );
rtTextureSampler<float4, 2> texture_diffuse1;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float2, texcoord, attribute texcoord, ); 

rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = optix::make_float3(0.0f);
	rtTerminateRay();
}

RT_PROGRAM void any_hit() {
  if (tex2D(texture_diffuse1, texcoord.x, texcoord.y).w < 0.1f) {
    rtIgnoreIntersection();
  } 
}

RT_PROGRAM void closest_hit()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 Kd_val = make_float3(tex2D( texture_diffuse1, texcoord.x, texcoord.y ));
  //float3 Kd_val = make_float3(tex2D( texture_diffuse1, 0.5f, 0.5f ));
  prd.result = Kd_val;
  //printf("color %f %f %f   ",prd.result.x,prd.result.y,prd.result.z);
}


