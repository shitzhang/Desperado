#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "common.h"
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<float4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );



RT_PROGRAM void pinhole_camera()
{

  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd ) ;

  output_buffer[launch_index] = make_float4( prd.result , 1.0f );
  
}

RT_PROGRAM void exception()
{
  rtPrintExceptionDetails();
  output_buffer[launch_index] = make_float4( bad_color , 1.0f );
}



rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}
